#include "hip/hip_runtime.h"
/*
 * This file is a stand-alone executable developed for the
 * testing of the C++ interface to the RTE+RRTMGP radiation code.
 *
 * It is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This software is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this software.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <boost/algorithm/string.hpp>
#include <chrono>
#include <iomanip>
#include <hip/hip_runtime_api.h>

#include "Status.h"
#include "Netcdf_interface.h"
#include "Array.h"
#include "raytracer_kernels_bw.h"
#include "Radiation_solver_bw.h"
#include "Aerosol_optics_rt.h"
#include "Gas_concs.h"
#include "types.h"
#include "mem_pool_gpu.h"


void read_and_set_vmr(
        const std::string& gas_name, const int n_col_x, const int n_col_y, const int n_lay,
        const Netcdf_handle& input_nc, Gas_concs& gas_concs)
{
    const std::string vmr_gas_name = "vmr_" + gas_name;

    if (input_nc.variable_exists(vmr_gas_name))
    {
        std::map<std::string, int> dims = input_nc.get_variable_dimensions(vmr_gas_name);
        const int n_dims = dims.size();

        if (n_dims == 0)
        {
            gas_concs.set_vmr(gas_name, input_nc.get_variable<Float>(vmr_gas_name));
        }
        else if (n_dims == 1)
        {
            if (dims.at("lay") == n_lay)
                gas_concs.set_vmr(gas_name,
                        Array<Float,1>(input_nc.get_variable<Float>(vmr_gas_name, {n_lay}), {n_lay}));
            else
                throw std::runtime_error("Illegal dimensions of gas \"" + gas_name + "\" in input");
        }
        else if (n_dims == 3)
        {
            if (dims.at("lay") == n_lay && dims.at("y") == n_col_y && dims.at("x") == n_col_x)
                gas_concs.set_vmr(gas_name,
                        Array<Float,2>(input_nc.get_variable<Float>(vmr_gas_name, {n_lay, n_col_y, n_col_x}), {n_col_x * n_col_y, n_lay}));
            else
                throw std::runtime_error("Illegal dimensions of gas \"" + gas_name + "\" in input");
        }
    }
    else
    {
        Status::print_warning("Gas \"" + gas_name + "\" not available in input file.");
    }
}

void read_and_set_aer(
        const std::string& aerosol_name, const int n_col_x, const int n_col_y, const int n_lay,
        const Netcdf_handle& input_nc, Aerosol_concs& aerosol_concs)
{
    if (input_nc.variable_exists(aerosol_name))
    {
        std::map<std::string, int> dims = input_nc.get_variable_dimensions(aerosol_name);
        const int n_dims = dims.size();

        if (n_dims == 1)
        {
            if (dims.at("lay") == n_lay)
                aerosol_concs.set_vmr(aerosol_name,
                        Array<Float,1>(input_nc.get_variable<Float>(aerosol_name, {n_lay}), {n_lay}));
            else
                throw std::runtime_error("Illegal dimensions of \"" + aerosol_name + "\" in input");
        }
        else if (n_dims == 3)
        {
            if (dims.at("lay") == n_lay && dims.at("y") == n_col_y && dims.at("x") == n_col_x)
                aerosol_concs.set_vmr(aerosol_name,
                        Array<Float,2>(input_nc.get_variable<Float>(aerosol_name, {n_lay, n_col_y, n_col_x}), {n_col_x * n_col_y, n_lay}));
            else
                throw std::runtime_error("Illegal dimensions of \"" + aerosol_name + "\" in input");
        }
        else
            throw std::runtime_error("Illegal dimensions of \"" + aerosol_name + "\" in input");
    }
    else
    {
        throw std::runtime_error("Aerosol type \"" + aerosol_name + "\" not available in input file.");
    }
}

void configure_memory_pool(int nlays, int ncols, int nchunks, int ngpts, int nbnds)
{
    /* Heuristic way to set up memory pool queues */
    std::map<std::size_t, std::size_t> pool_queues = {
        {64, 20},
        {128, 20},
        {256, 10},
        {512, 10},
        {1024, 5},
        {2048, 5},
        {nchunks * ngpts * sizeof(Float), 16},
        {nchunks * nbnds * sizeof(Float), 16},
        {(nlays + 1) * ncols * sizeof(Float), 14},
        {(nlays + 1) * nchunks * sizeof(Float), 10},
        {(nlays + 1) * nchunks * nbnds * sizeof(Float), 4},
        {(nlays + 1) * nchunks * ngpts * sizeof(int)/2, 6},
        {(nlays + 1) * nchunks * sizeof(Float), 18}
    };
    #ifdef GPU_MEM_POOL
    Memory_pool_gpu::init_instance(pool_queues);
    #endif
}

bool parse_command_line_options(
        std::map<std::string, std::pair<bool, std::string>>& command_line_options,
        int& photons_per_pixel,
        int argc, char** argv)
{
    for (int i=1; i<argc; ++i)
    {
        std::string argument(argv[i]);
        boost::trim(argument);

        if (argument == "-h" || argument == "--help")
        {
            Status::print_message("Possible usage:");
            for (const auto& clo : command_line_options)
            {
                std::ostringstream ss;
                ss << std::left << std::setw(30) << ("--" + clo.first);
                ss << clo.second.second << std::endl;
                Status::print_message(ss);
            }
            return true;
        }

        //check if option is integer n
        if (std::isdigit(argument[0]))
        {
            if (argument.size() > 1)
            {
                for (int i=1; i<argument.size(); ++i)
                {
                    if (!std::isdigit(argument[i]))
                    {
                        std::string error = argument + " is an illegal command line option.";
                        throw std::runtime_error(error);
                    }

                }
            }
            photons_per_pixel = int(std::stoi(argv[i]));
        }
        else
        {
            // Check if option starts with --
            if (argument[0] != '-' || argument[1] != '-')
            {
                std::string error = argument + " is an illegal command line option.";
                throw std::runtime_error(error);
            }
            else
                argument.erase(0, 2);

            // Check if option has prefix no-
            bool enable = true;
            if (argument[0] == 'n' && argument[1] == 'o' && argument[2] == '-')
            {
                enable = false;
                argument.erase(0, 3);
            }

            if (command_line_options.find(argument) == command_line_options.end())
            {
                std::string error = argument + " is an illegal command line option.";
                throw std::runtime_error(error);
            }
            else
                command_line_options.at(argument).first = enable;
        }
    }

    return false;
}


void print_command_line_options(
        const std::map<std::string, std::pair<bool, std::string>>& command_line_options)
{
    Status::print_message("Solver settings:");
    for (const auto& option : command_line_options)
    {
        std::ostringstream ss;
        ss << std::left << std::setw(20) << (option.first);
        ss << " = " << std::boolalpha << option.second.first << std::endl;
        Status::print_message(ss);
    }
}



void solve_radiation(int argc, char** argv)
{
    Status::print_message("###### Starting RTE+RRTMGP solver ######");

    ////// FLOW CONTROL SWITCHES //////
    // Parse the command line options.
    std::map<std::string, std::pair<bool, std::string>> command_line_options {
        {"shortwave"        , { true,  "Enable computation of shortwave radiation."  }},
        {"longwave"         , { false, "Enable computation of longwave radiation."   }},
        {"fluxes"           , { true,  "Enable computation of fluxes."               }},
        {"raytracing"       , { true,  "Use raytracing for flux computation."        }},
        {"cloud-optics"     , { false, "Enable cloud optics (both liquid and ice)."  }},
        {"liq-cloud-optics" , { false, "liquid only cloud optics."                   }},
        {"ice-cloud-optics" , { false, "ice only cloud optics."                      }},
        {"cloud-mie"        , { false, "mie cloud droplet scattering."               }},
        {"aerosol-optics"   , { false, "Enable aerosol optics."                      }},
        {"output-optical"   , { false, "Enable output of optical properties."        }},
        {"output-bnd-fluxes", { false, "Enable output of band fluxes."               }},
        {"lu-albedo"        , { false, "Compute spectral albedo from land use map"   }},
        {"image"            , { true,  "Compute XYZ values to generate RGB images"   }},
        {"broadband"        , { false, "Compute broadband radiances"                 }},
        {"profiling"        , { false, "Perform additional profiling run."           }},
        {"delta-cloud"      , { false, "delta-scaling of cloud optical properties"   }},
        {"delta-aerosol"    , { false, "delta-scaling of aerosol optical properties" }},
        {"cloud-cam"        , { false, "accumulate cloud water & ice paths for each camera pixel" }}};
    int photons_per_pixel = 1;

    if (parse_command_line_options(command_line_options, photons_per_pixel, argc, argv))
        return;


    const bool switch_shortwave         = command_line_options.at("shortwave"        ).first;
    const bool switch_longwave          = command_line_options.at("longwave"         ).first;
    const bool switch_fluxes            = command_line_options.at("fluxes"           ).first;
    bool switch_cloud_optics            = command_line_options.at("cloud-optics"     ).first;
    bool switch_liq_cloud_optics        = command_line_options.at("liq-cloud-optics" ).first;
    bool switch_ice_cloud_optics        = command_line_options.at("ice-cloud-optics" ).first;
    const bool switch_cloud_mie         = command_line_options.at("cloud-mie"        ).first;
    const bool switch_aerosol_optics    = command_line_options.at("aerosol-optics"   ).first;
    const bool switch_output_optical    = command_line_options.at("output-optical"   ).first;
    const bool switch_output_bnd_fluxes = command_line_options.at("output-bnd-fluxes").first;
    const bool switch_lu_albedo         = command_line_options.at("lu-albedo"        ).first;
    const bool switch_image             = command_line_options.at("image"            ).first;
    const bool switch_broadband         = command_line_options.at("broadband"        ).first;
    const bool switch_profiling         = command_line_options.at("profiling"        ).first;
    const bool switch_delta_cloud       = command_line_options.at("delta-cloud"      ).first;
    const bool switch_delta_aerosol     = command_line_options.at("delta-aerosol"    ).first;
    const bool switch_cloud_cam         = command_line_options.at("cloud-cam"        ).first;
    const bool switch_raytracing        = command_line_options.at("raytracing"       ).first;

    if (switch_longwave)
    {
        std::string error = "No longwave radiation implemented in the ray tracer";
        throw std::runtime_error(error);
    }

    if (switch_cloud_optics)
    {
        switch_liq_cloud_optics = true;
        switch_ice_cloud_optics = true;
    }
    if (switch_liq_cloud_optics || switch_ice_cloud_optics)
    {
        switch_cloud_optics = true;
    }

    if (switch_cloud_mie && switch_ice_cloud_optics)
    {
        std::string error = "Thou shall not use mie tables as long as ice optics are enabled";
        throw std::runtime_error(error);
    }

    // Print the options to the screen.
    print_command_line_options(command_line_options);

    Status::print_message("Using "+ std::to_string(photons_per_pixel) + " ray(s) per pixel");

    ////// READ THE ATMOSPHERIC DATA //////
    Status::print_message("Reading atmospheric input data from NetCDF.");

    Netcdf_file input_nc("rte_rrtmgp_input.nc", Netcdf_mode::Read);

    const int n_col_x = input_nc.get_dimension_size("x");
    const int n_col_y = input_nc.get_dimension_size("y");
    const int n_col = n_col_x * n_col_y;
    const int n_lay = input_nc.get_dimension_size("lay");
    const int n_lev = input_nc.get_dimension_size("lev");
    const int n_z = input_nc.get_dimension_size("z");

    // Read the x,y,z dimensions if raytracing is enabled
    Array<Float,1> grid_x(input_nc.get_variable<Float>("x", {n_col_x}), {n_col_x});
    Array<Float,1> grid_y(input_nc.get_variable<Float>("y", {n_col_y}), {n_col_y});
    Array<Float,1> grid_z(input_nc.get_variable<Float>("z", {n_z}), {n_z});
    Array<Float,1> z_lev(input_nc.get_variable<Float>("z_lev", {n_lev}), {n_lev});

    const Vector<int> grid_cells = {n_col_x, n_col_y, n_z};
    const Vector<Float> grid_d = {grid_x({2}) - grid_x({1}), grid_y({2}) - grid_y({1}), grid_z({2}) - grid_z({1})};
    const Vector<int> kn_grid = {input_nc.get_variable<int>("ngrid_x"),
                                 input_nc.get_variable<int>("ngrid_y"),
                                 input_nc.get_variable<int>("ngrid_z")};

    // Reading camera data
    Netcdf_group cam_in = input_nc.get_group("camera-settings");
    Camera camera;
    camera.fov    = cam_in.get_variable<Float>("fov");
    camera.cam_type = int(cam_in.get_variable<Float>("cam_type"));
    camera.position = {cam_in.get_variable<Float>("px"),
                       cam_in.get_variable<Float>("py"),
                       cam_in.get_variable<Float>("pz")};

    camera.nx  = int(cam_in.get_variable<Float>("nx"));
    camera.ny  = int(cam_in.get_variable<Float>("ny"));
    camera.npix = Int(camera.nx * camera.ny);

    camera.setup_rotation_matrix(cam_in.get_variable<Float>("yaw"),
                                 cam_in.get_variable<Float>("pitch"),
                                 cam_in.get_variable<Float>("roll"));
    camera.setup_normal_camera(camera);

    // Read the atmospheric fields.
    Array<Float,2> p_lay(input_nc.get_variable<Float>("p_lay", {n_lay, n_col_y, n_col_x}), {n_col, n_lay});
    Array<Float,2> t_lay(input_nc.get_variable<Float>("t_lay", {n_lay, n_col_y, n_col_x}), {n_col, n_lay});
    Array<Float,2> p_lev(input_nc.get_variable<Float>("p_lev", {n_lev, n_col_y, n_col_x}), {n_col, n_lev});
    Array<Float,2> t_lev(input_nc.get_variable<Float>("t_lev", {n_lev, n_col_y, n_col_x}), {n_col, n_lev});

    // read land use map if present, used for choosing between spectral and lambertian reflection and for spectral albedo
    // 0: water, 1: "grass", 2: "soil", 3: "concrete". Interpolating between 1 and 2 is currently possible
    Array<Float,1> land_use_map({n_col});
    if (input_nc.variable_exists("land_use_map") && switch_lu_albedo)
    {
        land_use_map = std::move(input_nc.get_variable<Float>("land_use_map", {n_col_y, n_col_x}));
    }
    else
    {
        // default to grass with some soil
        land_use_map.fill(Float(1.3));
    }

    // Fetch the col_dry in case present.
    Array<Float,2> col_dry;
    if (input_nc.variable_exists("col_dry"))
    {
        col_dry.set_dims({n_col, n_lay});
        col_dry = std::move(input_nc.get_variable<Float>("col_dry", {n_lay, n_col_y, n_col_x}));
    }

    // Create container for the gas concentrations and read gases.
    Gas_concs gas_concs;

    read_and_set_vmr("h2o", n_col_x, n_col_y, n_lay, input_nc, gas_concs);
    read_and_set_vmr("co2", n_col_x, n_col_y, n_lay, input_nc, gas_concs);
    read_and_set_vmr("o3" , n_col_x, n_col_y, n_lay, input_nc, gas_concs);
    read_and_set_vmr("n2o", n_col_x, n_col_y, n_lay, input_nc, gas_concs);
    read_and_set_vmr("co" , n_col_x, n_col_y, n_lay, input_nc, gas_concs);
    read_and_set_vmr("ch4", n_col_x, n_col_y, n_lay, input_nc, gas_concs);
    read_and_set_vmr("o2" , n_col_x, n_col_y, n_lay, input_nc, gas_concs);
    read_and_set_vmr("n2" , n_col_x, n_col_y, n_lay, input_nc, gas_concs);

    read_and_set_vmr("ccl4"   , n_col_x, n_col_y, n_lay, input_nc, gas_concs);
    read_and_set_vmr("cfc11"  , n_col_x, n_col_y, n_lay, input_nc, gas_concs);
    read_and_set_vmr("cfc12"  , n_col_x, n_col_y, n_lay, input_nc, gas_concs);
    read_and_set_vmr("cfc22"  , n_col_x, n_col_y, n_lay, input_nc, gas_concs);
    read_and_set_vmr("hfc143a", n_col_x, n_col_y, n_lay, input_nc, gas_concs);
    read_and_set_vmr("hfc125" , n_col_x, n_col_y, n_lay, input_nc, gas_concs);
    read_and_set_vmr("hfc23"  , n_col_x, n_col_y, n_lay, input_nc, gas_concs);
    read_and_set_vmr("hfc32"  , n_col_x, n_col_y, n_lay, input_nc, gas_concs);
    read_and_set_vmr("hfc134a", n_col_x, n_col_y, n_lay, input_nc, gas_concs);
    read_and_set_vmr("cf4"    , n_col_x, n_col_y, n_lay, input_nc, gas_concs);
    read_and_set_vmr("no2"    , n_col_x, n_col_y, n_lay, input_nc, gas_concs);

    Array<Float,2> lwp;
    Array<Float,2> iwp;
    Array<Float,2> rel;
    Array<Float,2> dei;

    if (switch_cloud_optics || switch_cloud_cam)
    {
        if (switch_liq_cloud_optics)
        {
            lwp.set_dims({n_col, n_lay});
            lwp = std::move(input_nc.get_variable<Float>("lwp", {n_lay, n_col_y, n_col_x}));

            rel.set_dims({n_col, n_lay});
            rel = std::move(input_nc.get_variable<Float>("rel", {n_lay, n_col_y, n_col_x}));
        }

        if (switch_ice_cloud_optics)
        {
            iwp.set_dims({n_col, n_lay});
            iwp = std::move(input_nc.get_variable<Float>("iwp", {n_lay, n_col_y, n_col_x}));

            dei.set_dims({n_col, n_lay});
            dei = std::move(input_nc.get_variable<Float>("dei", {n_lay, n_col_y, n_col_x}));
        }
    }
    else
    {
        rel.set_dims({n_col, n_lay});
        rel.fill(Float(0.));
    }

    Array<Float,2> rh;
    Aerosol_concs aerosol_concs;

    if (switch_aerosol_optics)
    {
        rh.set_dims({n_col, n_lay});
        rh = std::move(input_nc.get_variable<Float>("rh", {n_lay, n_col_y, n_col_x}));

        read_and_set_aer("aermr01", n_col_x, n_col_y, n_lay, input_nc, aerosol_concs);
        read_and_set_aer("aermr02", n_col_x, n_col_y, n_lay, input_nc, aerosol_concs);
        read_and_set_aer("aermr03", n_col_x, n_col_y, n_lay, input_nc, aerosol_concs);
        read_and_set_aer("aermr04", n_col_x, n_col_y, n_lay, input_nc, aerosol_concs);
        read_and_set_aer("aermr05", n_col_x, n_col_y, n_lay, input_nc, aerosol_concs);
        read_and_set_aer("aermr06", n_col_x, n_col_y, n_lay, input_nc, aerosol_concs);
        read_and_set_aer("aermr07", n_col_x, n_col_y, n_lay, input_nc, aerosol_concs);
        read_and_set_aer("aermr08", n_col_x, n_col_y, n_lay, input_nc, aerosol_concs);
        read_and_set_aer("aermr09", n_col_x, n_col_y, n_lay, input_nc, aerosol_concs);
        read_and_set_aer("aermr10", n_col_x, n_col_y, n_lay, input_nc, aerosol_concs);
        read_and_set_aer("aermr11", n_col_x, n_col_y, n_lay, input_nc, aerosol_concs);
    }



    ////// CREATE THE OUTPUT FILE //////
    // Create the general dimensions and arrays.
    Status::print_message("Preparing NetCDF output file.");

    Netcdf_file output_nc("rte_rrtmgp_output.nc", Netcdf_mode::Create);
    output_nc.add_dimension("x", camera.nx);
    output_nc.add_dimension("y", camera.ny);
    output_nc.add_dimension("pair", 2);

    int ngpts = 0;
    int nbnds = 0;
    if (switch_longwave)
    {
        Netcdf_file coef_nc_lw("coefficients_lw.nc", Netcdf_mode::Read);
        nbnds = std::max(coef_nc_lw.get_dimension_size("bnd"), nbnds);
        ngpts = std::max(coef_nc_lw.get_dimension_size("gpt"), ngpts);
    }
    if (switch_shortwave)
    {
        Netcdf_file coef_nc_sw("coefficients_sw.nc", Netcdf_mode::Read);
        nbnds = std::max(coef_nc_sw.get_dimension_size("bnd"), nbnds);
        ngpts = std::max(coef_nc_sw.get_dimension_size("gpt"), ngpts);
    }
    configure_memory_pool(n_lay, n_col, 1024, ngpts, nbnds);


/*    ////// RUN THE LONGWAVE SOLVER //////
    if (switch_longwave)
    {
        // Initialize the solver.
        Status::print_message("Initializing the longwave solver.");

        Gas_concs_gpu gas_concs_gpu(gas_concs);

        Radiation_solver_longwave rad_lw(gas_concs_gpu, "coefficients_lw.nc", "cloud_coefficients_lw.nc");

        // Read the boundary conditions.
        const int n_bnd_lw = rad_lw.get_n_bnd_gpu();
        const int n_gpt_lw = rad_lw.get_n_gpt_gpu();

        Array<Float,2> emis_sfc(input_nc.get_variable<Float>("emis_sfc", {n_col_y, n_col_x, n_bnd_lw}), {n_bnd_lw, n_col});
        Array<Float,1> t_sfc(input_nc.get_variable<Float>("t_sfc", {n_col_y, n_col_x}), {n_col});

        // Create output arrays.
        Array_gpu<Float,3> lw_tau;
        Array_gpu<Float,3> lay_source;
        Array_gpu<Float,3> lev_source_inc;
        Array_gpu<Float,3> lev_source_dec;
        Array_gpu<Float,2> sfc_source;

        if (switch_output_optical)
        {
            lw_tau        .set_dims({n_col, n_lay, n_gpt_lw});
            lay_source    .set_dims({n_col, n_lay, n_gpt_lw});
            lev_source_inc.set_dims({n_col, n_lay, n_gpt_lw});
            lev_source_dec.set_dims({n_col, n_lay, n_gpt_lw});
            sfc_source    .set_dims({n_col, n_gpt_lw});
        }

        Array_gpu<Float,2> lw_flux_up;
        Array_gpu<Float,2> lw_flux_dn;
        Array_gpu<Float,2> lw_flux_net;

        if (switch_fluxes)
        {
            lw_flux_up .set_dims({n_col, n_lev});
            lw_flux_dn .set_dims({n_col, n_lev});
            lw_flux_net.set_dims({n_col, n_lev});
        }

        Array_gpu<Float,3> lw_bnd_flux_up;
        Array_gpu<Float,3> lw_bnd_flux_dn;
        Array_gpu<Float,3> lw_bnd_flux_net;

        if (switch_output_bnd_fluxes)
        {
            lw_bnd_flux_up .set_dims({n_col, n_lev, n_bnd_lw});
            lw_bnd_flux_dn .set_dims({n_col, n_lev, n_bnd_lw});
            lw_bnd_flux_net.set_dims({n_col, n_lev, n_bnd_lw});
        }


        // Solve the radiation.

        Status::print_message("Solving the longwave radiation.");

        auto run_solver = [&]()
        {
            Array_gpu<Float,2> p_lay_gpu(p_lay);
            Array_gpu<Float,2> p_lev_gpu(p_lev);
            Array_gpu<Float,2> t_lay_gpu(t_lay);
            Array_gpu<Float,2> t_lev_gpu(t_lev);
            Array_gpu<Float,2> col_dry_gpu(col_dry);
            Array_gpu<Float,1> t_sfc_gpu(t_sfc);
            Array_gpu<Float,2> emis_sfc_gpu(emis_sfc);
            Array_gpu<Float,2> lwp_gpu(lwp);
            Array_gpu<Float,2> iwp_gpu(iwp);
            Array_gpu<Float,2> rel_gpu(rel);
            Array_gpu<Float,2> dei_gpu(dei);

            hipDeviceSynchronize();
            hipEvent_t start;
            hipEvent_t stop;
            hipEventCreate(&start);
            hipEventCreate(&stop);

            hipEventRecord(start, 0);

            rad_lw.solve_gpu(
                    switch_fluxes,
                    switch_cloud_optics,
                    switch_output_optical,
                    switch_output_bnd_fluxes,
                    gas_concs_gpu,
                    p_lay_gpu, p_lev_gpu,
                    t_lay_gpu, t_lev_gpu,
                    col_dry_gpu,
                    t_sfc_gpu, emis_sfc_gpu,
                    lwp_gpu, iwp_gpu,
                    rel_gpu, dei_gpu,
                    lw_tau, lay_source, lev_source_inc, lev_source_dec, sfc_source,
                    lw_flux_up, lw_flux_dn, lw_flux_net,
                    lw_bnd_flux_up, lw_bnd_flux_dn, lw_bnd_flux_net);

            hipEventRecord(stop, 0);
            hipEventSynchronize(stop);
            float duration = 0.f;
            hipEventElapsedTime(&duration, start, stop);

            hipEventDestroy(start);
            hipEventDestroy(stop);

            Status::print_message("Duration longwave solver: " + std::to_string(duration) + " (ms)");
        };

        // Tuning step;
        run_solver();

        // Profiling step;
        hipProfilerStart();
        run_solver();
        hipProfilerStop();

        constexpr int n_measures=10;
        for (int n=0; n<n_measures; ++n)
            run_solver();


        //// Store the output.
        Status::print_message("Storing the longwave output.");
        Array<Float,3> lw_tau_cpu(lw_tau);
        Array<Float,3> lay_source_cpu(lay_source);
        Array<Float,2> sfc_source_cpu(sfc_source);
        Array<Float,3> lev_source_inc_cpu(lev_source_inc);
        Array<Float,3> lev_source_dec_cpu(lev_source_dec);
        Array<Float,2> lw_flux_up_cpu(lw_flux_up);
        Array<Float,2> lw_flux_dn_cpu(lw_flux_dn);
        Array<Float,2> lw_flux_net_cpu(lw_flux_net);
        Array<Float,3> lw_bnd_flux_up_cpu(lw_bnd_flux_up);
        Array<Float,3> lw_bnd_flux_dn_cpu(lw_bnd_flux_dn);
        Array<Float,3> lw_bnd_flux_net_cpu(lw_bnd_flux_net);

        output_nc.add_dimension("gpt_lw", n_gpt_lw);
        output_nc.add_dimension("band_lw", n_bnd_lw);

        auto nc_lw_band_lims_wvn = output_nc.add_variable<Float>("lw_band_lims_wvn", {"band_lw", "pair"});
        nc_lw_band_lims_wvn.insert(rad_lw.get_band_lims_wavenumber_gpu().v(), {0, 0});

        if (switch_output_optical)
        {
            auto nc_lw_band_lims_gpt = output_nc.add_variable<int>("lw_band_lims_gpt", {"band_lw", "pair"});
            nc_lw_band_lims_gpt.insert(rad_lw.get_band_lims_gpoint_gpu().v(), {0, 0});

            auto nc_lw_tau = output_nc.add_variable<Float>("lw_tau", {"gpt_lw", "lay", "y", "x"});
            nc_lw_tau.insert(lw_tau_cpu.v(), {0, 0, 0, 0});

            auto nc_lay_source     = output_nc.add_variable<Float>("lay_source"    , {"gpt_lw", "lay", "y", "x"});
            auto nc_lev_source_inc = output_nc.add_variable<Float>("lev_source_inc", {"gpt_lw", "lay", "y", "x"});
            auto nc_lev_source_dec = output_nc.add_variable<Float>("lev_source_dec", {"gpt_lw", "lay", "y", "x"});

            auto nc_sfc_source = output_nc.add_variable<Float>("sfc_source", {"gpt_lw", "y", "x"});

            nc_lay_source.insert    (lay_source_cpu.v()    , {0, 0, 0, 0});
            nc_lev_source_inc.insert(lev_source_inc_cpu.v(), {0, 0, 0, 0});
            nc_lev_source_dec.insert(lev_source_dec_cpu.v(), {0, 0, 0, 0});

            nc_sfc_source.insert(sfc_source_cpu.v(), {0, 0, 0});
        }

        if (switch_fluxes)
        {
            auto nc_lw_flux_up  = output_nc.add_variable<Float>("lw_flux_up" , {"lev", "y", "x"});
            auto nc_lw_flux_dn  = output_nc.add_variable<Float>("lw_flux_dn" , {"lev", "y", "x"});
            auto nc_lw_flux_net = output_nc.add_variable<Float>("lw_flux_net", {"lev", "y", "x"});

            nc_lw_flux_up .insert(lw_flux_up_cpu .v(), {0, 0, 0});
            nc_lw_flux_dn .insert(lw_flux_dn_cpu .v(), {0, 0, 0});
            nc_lw_flux_net.insert(lw_flux_net_cpu.v(), {0, 0, 0});

            if (switch_output_bnd_fluxes)
            {
                auto nc_lw_bnd_flux_up  = output_nc.add_variable<Float>("lw_bnd_flux_up" , {"band_lw", "lev", "y", "x"});
                auto nc_lw_bnd_flux_dn  = output_nc.add_variable<Float>("lw_bnd_flux_dn" , {"band_lw", "lev", "y", "x"});
                auto nc_lw_bnd_flux_net = output_nc.add_variable<Float>("lw_bnd_flux_net", {"band_lw", "lev", "y", "x"});

                nc_lw_bnd_flux_up .insert(lw_bnd_flux_up_cpu.v(), {0, 0, 0, 0});
                nc_lw_bnd_flux_dn .insert(lw_bnd_flux_dn_cpu.v(), {0, 0, 0, 0});
                nc_lw_bnd_flux_net.insert(lw_bnd_flux_net_cpu.v(), {0, 0, 0, 0});
            }
        }
    }
*/

    ////// RUN THE SHORTWAVE SOLVER //////
    if (switch_shortwave)
    {
        // Initialize the solver.
        Status::print_message("Initializing the shortwave solver.");


        Gas_concs_gpu gas_concs_gpu(gas_concs);
        Radiation_solver_shortwave rad_sw(gas_concs_gpu, "coefficients_sw.nc", "cloud_coefficients_sw.nc","aerosol_optics.nc");

        // Read the boundary conditions.
        const int n_bnd_sw = rad_sw.get_n_bnd_gpu();
        const int n_gpt_sw = rad_sw.get_n_gpt_gpu();

        Array<Float,1> mu0(input_nc.get_variable<Float>("mu0", {n_col_y, n_col_x}), {n_col});
        Array<Float,1> azi(input_nc.get_variable<Float>("azi", {n_col_y, n_col_x}), {n_col});

        Array<Float,2> sfc_alb(input_nc.get_variable<Float>("sfc_alb_dir", {n_col_y, n_col_x, n_bnd_sw}), {n_bnd_sw, n_col});

        Array<Float,1> tsi_scaling({n_col});
        if (input_nc.variable_exists("tsi"))
        {
            Array<Float,1> tsi(input_nc.get_variable<Float>("tsi", {n_col_y, n_col_x}), {n_col});
            const Float tsi_ref = rad_sw.get_tsi_gpu();
            for (int icol=1; icol<=n_col; ++icol)
                tsi_scaling({icol}) = tsi({icol}) / tsi_ref;
        }
        else if (input_nc.variable_exists("tsi_scaling"))
        {
            Float tsi_scaling_in = input_nc.get_variable<Float>("tsi_scaling");
            for (int icol=1; icol<=n_col; ++icol)
                tsi_scaling({icol}) = tsi_scaling_in;
        }
        else
        {
            for (int icol=1; icol<=n_col; ++icol)
                tsi_scaling({icol}) = Float(1.);
        }

        Array_gpu<Float,3> XYZ;
        Array_gpu<Float,2> radiance;

        if (switch_broadband)
        {
            radiance.set_dims({camera.nx, camera.ny});
        }
        if (switch_image)
        {
            XYZ.set_dims({camera.nx, camera.ny, 3});
        }

        if (switch_cloud_mie)
            rad_sw.load_mie_tables("mie_lut_broadband.nc", "mie_lut_visualisation.nc", switch_broadband, switch_image);


        Array_gpu<Float,2> liwp_cam;
        Array_gpu<Float,2> tauc_cam;
        Array_gpu<Float,2> dist_cam;
        Array_gpu<Float,2> zen_cam;

        if (switch_cloud_cam)
        {
            liwp_cam.set_dims({camera.nx, camera.ny});
            tauc_cam.set_dims({camera.nx, camera.ny});
            dist_cam.set_dims({camera.nx, camera.ny});
            zen_cam.set_dims({camera.nx, camera.ny});
        }

        // Solve the radiation.
        Status::print_message("Solving the shortwave radiation.");

        auto run_solver_bb = [&](const bool tune_step)
        {
            Array_gpu<Float,2> p_lay_gpu(p_lay);
            Array_gpu<Float,2> p_lev_gpu(p_lev);
            Array_gpu<Float,2> t_lay_gpu(t_lay);
            Array_gpu<Float,2> t_lev_gpu(t_lev);
            Array_gpu<Float,1> z_lev_gpu(z_lev);
            Array_gpu<Float,2> col_dry_gpu(col_dry);
            Array_gpu<Float,2> sfc_alb_gpu(sfc_alb);
            Array_gpu<Float,1> tsi_scaling_gpu(tsi_scaling);
            Array_gpu<Float,1> mu0_gpu(mu0);
            Array_gpu<Float,1> azi_gpu(azi);
            Array_gpu<Float,2> lwp_gpu(lwp);
            Array_gpu<Float,2> iwp_gpu(iwp);
            Array_gpu<Float,2> rel_gpu(rel);
            Array_gpu<Float,2> dei_gpu(dei);

            Array_gpu<Float,2> rh_gpu(rh);
            Aerosol_concs_gpu aerosol_concs_gpu(aerosol_concs);

            Array_gpu<Float,1> land_use_map_gpu(land_use_map);

            hipDeviceSynchronize();
            hipEvent_t start;
            hipEvent_t stop;
            hipEventCreate(&start);
            hipEventCreate(&stop);

            hipEventRecord(start, 0);

            rad_sw.solve_gpu_bb(
                    switch_cloud_optics,
                    switch_cloud_mie,
                    switch_aerosol_optics,
                    switch_lu_albedo,
                    switch_delta_cloud,
                    switch_delta_aerosol,
                    switch_cloud_cam,
                    switch_raytracing,
                    grid_cells,
                    grid_d,
                    kn_grid,
                    photons_per_pixel,
                    gas_concs_gpu,
                    p_lay_gpu, p_lev_gpu,
                    t_lay_gpu, t_lev_gpu,
                    z_lev_gpu,
                    col_dry_gpu,
                    sfc_alb_gpu,
                    tsi_scaling_gpu,
                    mu0_gpu, azi_gpu,
                    lwp_gpu, iwp_gpu,
                    rel_gpu, dei_gpu,
                    land_use_map_gpu,
                    rh_gpu,
                    aerosol_concs,
                    camera,
                    radiance,
                    liwp_cam,
                    tauc_cam,
                    dist_cam,
                    zen_cam);

            hipEventRecord(stop, 0);
            hipEventSynchronize(stop);
            float duration = 0.f;
            hipEventElapsedTime(&duration, start, stop);

            hipEventDestroy(start);
            hipEventDestroy(stop);

            Status::print_message("Duration shortwave solver (broadband version): " + std::to_string(duration) + " (ms)");
        };

        auto run_solver = [&](const bool tune_step)
        {
            Array_gpu<Float,2> p_lay_gpu(p_lay);
            Array_gpu<Float,2> p_lev_gpu(p_lev);
            Array_gpu<Float,2> t_lay_gpu(t_lay);
            Array_gpu<Float,2> t_lev_gpu(t_lev);
            Array_gpu<Float,1> z_lev_gpu(z_lev);
            Array_gpu<Float,2> col_dry_gpu(col_dry);
            Array_gpu<Float,2> sfc_alb_gpu(sfc_alb);
            Array_gpu<Float,1> tsi_scaling_gpu(tsi_scaling);
            Array_gpu<Float,1> mu0_gpu(mu0);
            Array_gpu<Float,1> azi_gpu(azi);
            Array_gpu<Float,2> lwp_gpu(lwp);
            Array_gpu<Float,2> iwp_gpu(iwp);
            Array_gpu<Float,2> rel_gpu(rel);
            Array_gpu<Float,2> dei_gpu(dei);

            Array_gpu<Float,2> rh_gpu(rh);
            Aerosol_concs_gpu aerosol_concs_gpu(aerosol_concs);

            Array_gpu<Float,1> land_use_map_gpu(land_use_map);

            hipDeviceSynchronize();
            hipEvent_t start;
            hipEvent_t stop;
            hipEventCreate(&start);
            hipEventCreate(&stop);

            hipEventRecord(start, 0);

            rad_sw.solve_gpu(
                    tune_step,
                    switch_cloud_optics,
                    switch_cloud_mie,
                    switch_aerosol_optics,
                    switch_lu_albedo,
                    switch_delta_cloud,
                    switch_delta_aerosol,
                    switch_cloud_cam,
                    switch_raytracing,
                    grid_cells,
                    grid_d,
                    kn_grid,
                    photons_per_pixel,
                    gas_concs_gpu,
                    p_lay_gpu, p_lev_gpu,
                    t_lay_gpu, t_lev_gpu,
                    z_lev_gpu,
                    col_dry_gpu,
                    sfc_alb_gpu,
                    tsi_scaling_gpu,
                    mu0_gpu, azi_gpu,
                    lwp_gpu, iwp_gpu,
                    rel_gpu, dei_gpu,
                    land_use_map_gpu,
                    rh_gpu,
                    aerosol_concs,
                    camera,
                    XYZ,
                    liwp_cam,
                    tauc_cam,
                    dist_cam,
                    zen_cam);

            hipEventRecord(stop, 0);
            hipEventSynchronize(stop);
            float duration = 0.f;
            hipEventElapsedTime(&duration, start, stop);

            hipEventDestroy(start);
            hipEventDestroy(stop);

            Status::print_message("Duration shortwave solver (image version): " + std::to_string(duration) + " (ms)");
        };

        if (switch_broadband)
        {
           // Profiling step;
           run_solver_bb(false);

           if (switch_profiling)
           {
               hipProfilerStart();
               run_solver_bb(false);
               hipProfilerStop();
            }
        }
        if (switch_image)
        {
            // tune step
            run_solver(true);

            // actual solve
            run_solver(false);

            // Profiling step;
            if (switch_profiling)
            {
                hipProfilerStart();
                run_solver(false);
                hipProfilerStop();
            }
        }

        // Store the output.
        Status::print_message("Storing the shortwave output.");

        if (switch_raytracing)
        {
            output_nc.add_dimension("gpt_sw", n_gpt_sw);
            output_nc.add_dimension("band_sw", n_bnd_sw);

            auto nc_sw_band_lims_wvn = output_nc.add_variable<Float>("sw_band_lims_wvn", {"band_sw", "pair"});
            nc_sw_band_lims_wvn.insert(rad_sw.get_band_lims_wavenumber_gpu().v(), {0, 0});

            if (switch_broadband)
            {
                Array<Float,2> radiance_cpu(radiance);

                auto nc_var = output_nc.add_variable<Float>("radiance", {"y","x"});
                nc_var.insert(radiance_cpu.v(), {0, 0});
                nc_var.add_attribute("long_name", "shortwave radiance");
                nc_var.add_attribute("units", "W m-2 sr-1");
            }
            if (switch_image)
            {
                Array<Float,3> xyz_cpu(XYZ);
                output_nc.add_dimension("n",3);

                auto nc_xyz = output_nc.add_variable<Float>("XYZ", {"n","y","x"});
                nc_xyz.insert(xyz_cpu.v(), {0, 0, 0});

                nc_xyz.add_attribute("long_name", "X Y Z tristimulus values");
            }
        }

        if (switch_cloud_cam)
        {
            Array<Float,2> liwp_cam_cpu(liwp_cam);
            Array<Float,2> tauc_cam_cpu(tauc_cam);
            Array<Float,2> dist_cam_cpu(dist_cam);
            Array<Float,2> zen_cam_cpu(zen_cam);

            auto nc_var_liwp = output_nc.add_variable<Float>("liq_ice_wp_cam", {"y","x"});
            nc_var_liwp.insert(liwp_cam_cpu.v(), {0, 0});
            nc_var_liwp.add_attribute("long_name", "accumulated liquid+ice water path");

            auto nc_var_tauc = output_nc.add_variable<Float>("tau_cld_cam", {"y","x"});
            nc_var_tauc.insert(tauc_cam_cpu.v(), {0, 0});
            nc_var_tauc.add_attribute("long_name", "accumulated cloud optical depth (441-615nm band)");

            auto nc_var_dist = output_nc.add_variable<Float>("dist_cld_cam", {"y","x"});
            nc_var_dist.insert(dist_cam_cpu.v(), {0, 0});
            nc_var_dist.add_attribute("long_name", "distance to first cloudy cell");

            auto nc_var_csza = output_nc.add_variable<Float>("zen_cam", {"y","x"});
            nc_var_csza.insert(zen_cam_cpu.v(), {0, 0});
            nc_var_csza.add_attribute("long_name", "zenith angle of camera pixel");
        }

        auto nc_mu0 = output_nc.add_variable<Float>("sza");
        nc_mu0.insert(acos(mu0({1}))/M_PI * Float(180.), {0});

        auto nc_azi = output_nc.add_variable<Float>("azi");
        nc_azi.insert(azi({1})/M_PI * Float(180.), {0});

        // camera position and direction
        Netcdf_group output_cam = output_nc.add_group("camera-settings");

        std::string cam_vars[] = {"yaw","pitch","roll","px","py","pz"};
        for (auto &&cam_var : cam_vars)
        {
            auto nc_cam_out = output_cam.add_variable<Float>(cam_var);
            nc_cam_out.insert(cam_in.get_variable<Float>(cam_var), {0});
        }
    }

    Status::print_message("###### Finished RTE+RRTMGP solver ######");
}


int main(int argc, char** argv)
{
    try
    {
        solve_radiation(argc, argv);
    }

    // Catch any exceptions and return 1.
    catch (const std::exception& e)
    {
        std::string error = "EXCEPTION: " + std::string(e.what());
        Status::print_message(error);
        return 1;
    }
    catch (...)
    {
        Status::print_message("UNHANDLED EXCEPTION!");
        return 1;
    }

    // Return 0 in case of normal exit.
    return 0;
}
